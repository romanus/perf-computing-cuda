#include "hip/hip_runtime.h"
#include "main.cuh"

using float32_t = float;

constexpr auto KERNEL_RADIUS = 2;
constexpr auto KERNEL_W = (2 * KERNEL_RADIUS + 1); // 5 x 5 kernel

__device__ __constant__ float32_t d_Kernel[KERNEL_W];

constexpr auto TILE_W = 16;		// active cell width
constexpr auto TILE_H = 16;		// active cell height

#define IMUL(a,b) __mul24(a,b)

__global__ void convolutionRowGPU(float32_t* d_Result, const float32_t* d_Data, int dataW, int dataH)
{
	// Data cache: threadIdx.x , threadIdx.y
	__shared__ float32_t data[3 * TILE_H * (TILE_W + KERNEL_RADIUS * 2)]; // 3 channels of TILE_H rows and (TILE_W + KERNEL_RADIUS * 2) columns

	// original image based coordinate
	const int x0 = threadIdx.x + IMUL(blockIdx.x, blockDim.x);
	const int shift = threadIdx.y * (TILE_W + KERNEL_RADIUS * 2);

	const int gLoc = threadIdx.x +
		IMUL(blockIdx.x, blockDim.x) +
		IMUL(threadIdx.y, dataW) +
		IMUL(blockIdx.y, blockDim.y) * dataW;

	// load cache (32x16 shared memory, 16x16 threads blocks)
	// each threads loads two values from global memory into shared mem
	// if in image area, get value in global mem, else 0
	int x;		// image based coordinate

	// case1: left
	x = x0 - KERNEL_RADIUS;
	if (x < 0)
	{
		data[3 * (threadIdx.x + shift) + 0] = 0;
		data[3 * (threadIdx.x + shift) + 1] = 0;
		data[3 * (threadIdx.x + shift) + 2] = 0;
	}
	else
	{
		data[3 * (threadIdx.x + shift) + 0] = d_Data[3 * (gLoc - KERNEL_RADIUS) + 0];
		data[3 * (threadIdx.x + shift) + 1] = d_Data[3 * (gLoc - KERNEL_RADIUS) + 1];
		data[3 * (threadIdx.x + shift) + 2] = d_Data[3 * (gLoc - KERNEL_RADIUS) + 2];
	}

	// case2: right
	x = x0 + KERNEL_RADIUS;
	if (x > dataW - 1)
	{
		data[3 * (threadIdx.x + 2 * KERNEL_RADIUS + shift) + 0] = 0;
		data[3 * (threadIdx.x + 2 * KERNEL_RADIUS + shift) + 1] = 0;
		data[3 * (threadIdx.x + 2 * KERNEL_RADIUS + shift) + 2] = 0;
	}
	else
	{
		data[3 * (threadIdx.x + 2 * KERNEL_RADIUS + shift) + 0] = d_Data[3 * (gLoc + KERNEL_RADIUS) + 0];
		data[3 * (threadIdx.x + 2 * KERNEL_RADIUS + shift) + 1] = d_Data[3 * (gLoc + KERNEL_RADIUS) + 1];
		data[3 * (threadIdx.x + 2 * KERNEL_RADIUS + shift) + 2] = d_Data[3 * (gLoc + KERNEL_RADIUS) + 2];
	}

	data[3 * (threadIdx.x + KERNEL_RADIUS + shift) + 0] = d_Data[3 * gLoc + 0];
	data[3 * (threadIdx.x + KERNEL_RADIUS + shift) + 1] = d_Data[3 * gLoc + 1];
	data[3 * (threadIdx.x + KERNEL_RADIUS + shift) + 2] = d_Data[3 * gLoc + 2];

	__syncthreads();

	// convolution
	float32_t sum[] = { 0, 0, 0 };
	x = threadIdx.x + KERNEL_RADIUS;
	for (int i = -KERNEL_RADIUS; i <= KERNEL_RADIUS; i++)
	{
		sum[0] += data[3 * (x + i + shift) + 0] * d_Kernel[i + KERNEL_RADIUS];
		sum[1] += data[3 * (x + i + shift) + 1] * d_Kernel[i + KERNEL_RADIUS];
		sum[2] += data[3 * (x + i + shift) + 2] * d_Kernel[i + KERNEL_RADIUS];
	}

	d_Result[3 * gLoc + 0] = sum[0];
	d_Result[3 * gLoc + 1] = sum[1];
	d_Result[3 * gLoc + 2] = sum[2];
}

__global__ void convolutionColGPU(float32_t* d_Result, const float32_t* d_Data, int dataW, int dataH)
{
	// Data cache: threadIdx.x , threadIdx.y
	__shared__ float32_t data[3 * TILE_W * (TILE_H + KERNEL_RADIUS * 2)]; // 3 channels of (TILE_H + KERNEL_RADIUS * 2) rows and TILE_W columns

	// original image based coordinate
	const int y0 = threadIdx.y + IMUL(blockIdx.y, blockDim.y);
	const int shift = threadIdx.y * TILE_W;

	// global mem address of this thread
	const int gLoc = threadIdx.x +
		IMUL(blockIdx.x, blockDim.x) +
		IMUL(threadIdx.y, dataW) +
		IMUL(blockIdx.y, blockDim.y) * dataW;

	// load cache (32x16 shared memory, 16x16 threads blocks)
	// each threads loads two values from global memory into shared mem
	// if in image area, get value in global mem, else 0
	int y;		// image based coordinate

	// case1: upper
	y = y0 - KERNEL_RADIUS;
	if (y < 0)
	{
		data[3 * (threadIdx.x + shift) + 0] = 0;
		data[3 * (threadIdx.x + shift) + 1] = 0;
		data[3 * (threadIdx.x + shift) + 2] = 0;
	}
	else
	{
		data[3 * (threadIdx.x + shift) + 0] = d_Data[3 * (gLoc - IMUL(dataW, KERNEL_RADIUS)) + 0];
		data[3 * (threadIdx.x + shift) + 1] = d_Data[3 * (gLoc - IMUL(dataW, KERNEL_RADIUS)) + 1];
		data[3 * (threadIdx.x + shift) + 2] = d_Data[3 * (gLoc - IMUL(dataW, KERNEL_RADIUS)) + 2];
	}

	// case2: lower
	y = y0 + KERNEL_RADIUS;
	const auto shift1 = shift + IMUL(2 * KERNEL_RADIUS, TILE_W);
	if (y > dataH - 1)
	{
		data[3 * (threadIdx.x + shift1) + 0] = 0;
		data[3 * (threadIdx.x + shift1) + 1] = 0;
		data[3 * (threadIdx.x + shift1) + 2] = 0;
	}
	else
	{
		data[3 * (threadIdx.x + shift1) + 0] = d_Data[3 * (gLoc + IMUL(dataW, KERNEL_RADIUS)) + 0];
		data[3 * (threadIdx.x + shift1) + 1] = d_Data[3 * (gLoc + IMUL(dataW, KERNEL_RADIUS)) + 1];
		data[3 * (threadIdx.x + shift1) + 2] = d_Data[3 * (gLoc + IMUL(dataW, KERNEL_RADIUS)) + 2];
	}

	data[3 * (threadIdx.x + shift + IMUL(TILE_W, KERNEL_RADIUS)) + 0] = d_Data[3 * gLoc + 0];
	data[3 * (threadIdx.x + shift + IMUL(TILE_W, KERNEL_RADIUS)) + 1] = d_Data[3 * gLoc + 1];
	data[3 * (threadIdx.x + shift + IMUL(TILE_W, KERNEL_RADIUS)) + 2] = d_Data[3 * gLoc + 2];

	__syncthreads();

	// convolution
	float32_t sum[] = { 0, 0, 0 };
	for (int i = 0; i <= KERNEL_RADIUS * 2; i++)
	{
		sum[0] += data[3 * (threadIdx.x + (threadIdx.y + i) * TILE_W) + 0] * d_Kernel[i];
		sum[1] += data[3 * (threadIdx.x + (threadIdx.y + i) * TILE_W) + 1] * d_Kernel[i];
		sum[2] += data[3 * (threadIdx.x + (threadIdx.y + i) * TILE_W) + 2] * d_Kernel[i];
	}

	d_Result[3 * gLoc + 0] = sum[0];
	d_Result[3 * gLoc + 1] = sum[1];
	d_Result[3 * gLoc + 2] = sum[2];
}

constexpr auto KERNEL_SIZE = static_cast<int32_t>(KERNEL_W * sizeof(float32_t));

void FilterBenchmark(const cv::Mat& image)
{
	std::cout << "----------- CONVOLUTION ------------\n";

	const auto h_Kernel = cv::getGaussianKernel(KERNEL_W, -1, CV_32F);
	hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), h_Kernel.data, KERNEL_SIZE);

	auto multiplier = size_t{};
	DeviceAlloc::ComputeSize(image, &multiplier);

	const auto dw = image.cols;
	const auto dh = cv::saturate_cast<int>(image.rows * multiplier);

	auto h_Result = cv::Mat(dh, dw, CV_32FC3);

	dim3 blocks(TILE_W, TILE_H);
	dim3 grids(dw / TILE_W, dh / TILE_H); // we assume that image width and height divide by TILE_W/TILE_H

	{
		const auto timeLock = MeasureTime("Time computing+load+unload");

		const auto d_Image = DeviceAlloc(image);

		auto d_Result = DeviceAlloc(image);
		auto d_Data = DeviceAlloc(d_Image.m_size);

		{
			const auto timeLock = MeasureTime("Time computing+unload");

			{
				const auto timeLock3 = MeasureTime("Time computing");
				convolutionRowGPU<<<grids, blocks>>>((float32_t*)d_Data.m_deviceData, (const float32_t*)d_Image.m_deviceData, dw, dh);
				convolutionColGPU<<<grids, blocks>>>((float32_t*)d_Result.m_deviceData, (const float32_t*)d_Data.m_deviceData, dw, dh);
			}

			hipDeviceSynchronize();
			d_Result.CopyToHost(h_Result.data);
		}

		hipDeviceSynchronize();
	}

	// compare the output and OpenCV output
	
	auto openCV_input = cv::Mat(dh, dw, CV_32FC3);
	DeviceAlloc(image).CopyToHost(openCV_input.data);
	auto openCV_output = cv::Mat{};
	cv::sepFilter2D(openCV_input, openCV_output, CV_32F, h_Kernel, h_Kernel, cv::Point(-1, -1), 0, cv::BorderTypes::BORDER_CONSTANT);

	const auto algoOutputEqual = std::equal(h_Result.datastart, h_Result.dataend, openCV_output.datastart);
	std::cout << "The CUDA algorithm matches OpenCV's algo: " << std::boolalpha << algoOutputEqual << std::endl;

    std::cout << "------------------------------------\n" << std::endl;
}