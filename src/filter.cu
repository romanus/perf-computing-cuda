#include "hip/hip_runtime.h"
#include "main.cuh"

constexpr auto KERNEL_RADIUS = 8;
constexpr auto KERNEL_W = (2 * KERNEL_RADIUS + 1);

__device__ __constant__ float d_Kernel[KERNEL_W];

constexpr auto TILE_W = 16;		// active cell width
constexpr auto TILE_H = 16;		// active cell height
constexpr auto TILE_SIZE = (TILE_W + KERNEL_RADIUS * 2) * (TILE_W + KERNEL_RADIUS * 2);

#define IMUL(a,b) __mul24(a,b)

__global__ void convolutionRowGPU(float* d_Result, float* d_Data, int dataW, int dataH)
{
	// Data cache: threadIdx.x , threadIdx.y
	__shared__ float data[TILE_H * (TILE_W + KERNEL_RADIUS * 2)];

	// global mem address of this thread
	const int gLoc = threadIdx.x +
		IMUL(blockIdx.x, blockDim.x) +
		IMUL(threadIdx.y, dataW) +
		IMUL(blockIdx.y, blockDim.y) * dataW;

	// load cache (32x16 shared memory, 16x16 threads blocks)
	// each threads loads two values from global memory into shared mem
	// if in image area, get value in global mem, else 0
	int x;		// image based coordinate

	// original image based coordinate
	const int x0 = threadIdx.x + IMUL(blockIdx.x, blockDim.x);
	const int shift = threadIdx.y * (TILE_W + KERNEL_RADIUS * 2);

	// case1: left
	x = x0 - KERNEL_RADIUS;
	if (x < 0)
		data[threadIdx.x + shift] = 0;
	else
		data[threadIdx.x + shift] = d_Data[gLoc - KERNEL_RADIUS];

	// case2: right
	x = x0 + KERNEL_RADIUS;
	if (x > dataW - 1)
		data[threadIdx.x + blockDim.x + shift] = 0;
	else
		data[threadIdx.x + blockDim.x + shift] = d_Data[gLoc + KERNEL_RADIUS];

	__syncthreads();

	// convolution
	float sum = 0;
	x = KERNEL_RADIUS + threadIdx.x;
	for (int i = -KERNEL_RADIUS; i <= KERNEL_RADIUS; i++)
		sum += data[x + i + shift] * d_Kernel[KERNEL_RADIUS + i];

	d_Result[gLoc] = sum;

}

__global__ void convolutionColGPU(float* d_Result, float* d_Data, int dataW, int dataH)
{
	// Data cache: threadIdx.x , threadIdx.y
	__shared__ float data[TILE_W * (TILE_H + KERNEL_RADIUS * 2)];

	// global mem address of this thread
	const int gLoc = threadIdx.x +
		IMUL(blockIdx.x, blockDim.x) +
		IMUL(threadIdx.y, dataW) +
		IMUL(blockIdx.y, blockDim.y) * dataW;

	// load cache (32x16 shared memory, 16x16 threads blocks)
	// each threads loads two values from global memory into shared mem
	// if in image area, get value in global mem, else 0
	int y;		// image based coordinate

	// original image based coordinate
	const int y0 = threadIdx.y + IMUL(blockIdx.y, blockDim.y);
	const int shift = threadIdx.y * (TILE_W);

	// case1: upper
	y = y0 - KERNEL_RADIUS;
	if (y < 0)
		data[threadIdx.x + shift] = 0;
	else
		data[threadIdx.x + shift] = d_Data[gLoc - IMUL(dataW, KERNEL_RADIUS)];

	// case2: lower
	y = y0 + KERNEL_RADIUS;
	const int shift1 = shift + IMUL(blockDim.y, TILE_W);
	if (y > dataH - 1)
		data[threadIdx.x + shift1] = 0;
	else
		data[threadIdx.x + shift1] = d_Data[gLoc + IMUL(dataW, KERNEL_RADIUS)];

	__syncthreads();

	// convolution
	float sum = 0;
	for (int i = 0; i <= KERNEL_RADIUS * 2; i++)
		sum += data[threadIdx.x + (threadIdx.y + i) * TILE_W] * d_Kernel[i];

	d_Result[gLoc] = sum;

}

//Image width should be aligned to maximum coalesced read/write size
//for best global memory performance in both row and column filter.
constexpr auto KERNEL_SIZE = static_cast<int>(KERNEL_W * sizeof(float));

void FilterBenchmark(const cv::Mat& image)
{
    std::cout << "----------- CONVOLUTION ------------\n";

	float* h_Kernel;
	float* h_DataR, * h_DataG, * h_DataB, * h_ResultR, * h_ResultG, * h_ResultB;
	float* d_DataA, * d_DataB;

	double gpuTime, runTime, singleRunTime;

	int i, dw, dh, data_size;
	dw = dh = 1024;

	data_size = dw * dh * sizeof(int);

	h_Kernel = (float*)malloc(KERNEL_SIZE);

	h_DataR = (float*)malloc(data_size);
	h_DataG = (float*)malloc(data_size);
	h_DataB = (float*)malloc(data_size);
	h_ResultR = (float*)malloc(data_size);
	h_ResultG = (float*)malloc(data_size);
	h_ResultB = (float*)malloc(data_size);

	hipMalloc((void**)&d_DataA, data_size);
	hipMalloc((void**)&d_DataB, data_size);

	// initialize kernel
	float kernelSum = 0;
	for (i = 0; i < KERNEL_W; i++) {
		float dist = (float)(i - KERNEL_RADIUS) / (float)KERNEL_RADIUS;
		h_Kernel[i] = expf(-dist * dist / 2);
		kernelSum += h_Kernel[i];
	}
	for (i = 0; i < KERNEL_W; i++)
		h_Kernel[i] /= kernelSum;

	// loadRawImage(iFilename, dw, dh, h_DataR, h_DataG, h_DataB))

	hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), h_Kernel, KERNEL_SIZE);

	dim3 blocks(TILE_W, TILE_H);
	dim3 grids(dw / TILE_W, dh / TILE_H);

	// red channel
	hipMemcpy(d_DataA, h_DataR, data_size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	convolutionRowGPU<<<grids, blocks>>>(d_DataB, d_DataA, dw, dh);
	convolutionColGPU<<<grids, blocks>>>(d_DataA, d_DataB, dw, dh);
	hipDeviceSynchronize();

	// read back GPU result
	hipMemcpy(h_ResultR, d_DataA, data_size, hipMemcpyDeviceToHost);

	// green channel
	hipMemcpy(d_DataA, h_DataG, data_size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	convolutionRowGPU<<<grids, blocks>>>(d_DataB, d_DataA, dw, dh);
	convolutionColGPU<<<grids, blocks>>>(d_DataA, d_DataB, dw, dh);
	hipDeviceSynchronize();

	// read back GPU result
	hipMemcpy(h_ResultG, d_DataA, data_size, hipMemcpyDeviceToHost);

	// blue channel
	hipMemcpy(d_DataA, h_DataB, data_size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	convolutionRowGPU<<<grids, blocks>>>(d_DataB, d_DataA, dw, dh);
	convolutionColGPU<<<grids, blocks>>>(d_DataA, d_DataB, dw, dh);
	hipDeviceSynchronize();

	// read back GPU result
	hipMemcpy(h_ResultB, d_DataA, data_size, hipMemcpyDeviceToHost);

	hipFree(d_DataB);
	hipFree(d_DataA);

	free(h_ResultB);
	free(h_ResultG);
	free(h_ResultR);
	free(h_DataB);
	free(h_DataG);
	free(h_DataR);
	free(h_Kernel);

    std::cout << "------------------------------------\n" << std::endl;
}